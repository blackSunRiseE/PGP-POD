
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void kernel(void){
   printf("Hello World!\n");
}

int main() {
    kernel<<< 1, 10>>>();
    hipDeviceSynchronize(); 
    return 0;
}